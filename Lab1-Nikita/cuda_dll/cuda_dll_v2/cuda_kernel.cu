#include "hip/hip_runtime.h"
#include <windows.h>
#include "hip/hip_runtime.h"
#include ""
#include "cuda_dll_header.h"


hipError_t addWithCuda(float * a, float * b, float * c, int N);
hipError_t mulWithCuda(float * a, float * b, float * c, int N);
hipError_t transWithCuda(float * a, float * b, int N,int M);
hipError_t matMulWithCuda(float * a, float * b, float * c, int M, int K, int N);

//����� ��������
__global__ void addKernel(float *a, float *b, float *c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
int sum(char* s1, char* s2, char* s3, float * a, float * b, float * c, int N)
{
	char* s11; char* s22; char* s33;
	s11 = s1; s22 = s2; s33 = s3;
	while (*s11) { *s33 = *s11; s33++; s11++; }
	while (*s22) { *s33 = *s22; s33++; s22++; }
	*s33 = 0;

	hipError_t cudaStatus = addWithCuda(a, b, c, N);
	cudaStatus = hipDeviceReset();
	return 0;
}
hipError_t addWithCuda(float * a, float * b, float * c, int N)
{
	int size = N * sizeof(float);
	hipError_t cudaStatus;
	/*Define and initialize arrays in HOST*/
	float* h_A = (float *)malloc(size);
	float* h_B = (float *)malloc(size);
	float* h_C = (float *)malloc(size);

	for (int i = 0; i < N; i++) {
		h_A[i] = a[i];
		h_B[i] = b[i];
		h_C[i] = c[i];
	}

	/*Define and allocate arrays in DEVICE*/
	float* d_A;
	float* d_B;
	float* d_C;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	/*Copy arrays from HOST to DEVICE*/
	cudaStatus = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

	/*Define level of parallelism*/
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(1, 1, 1);

	/*Launch kernel and synchronize*/
	addKernel << <numBlocks, threadsPerBlock >> > (d_A, d_B, d_C);
	cudaStatus = hipDeviceSynchronize();

	/*Copy output array from DEVICE to HOST*/
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		c[i] = h_C[i];
	}
	/*Free device memory*/
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return cudaStatus;
}

//������.��������� ��������
__global__ void mulKernel(float *a, float *b, float *c)
{
	int i = threadIdx.x;
	c[i] = a[i] * b[i];
}
int mul(char* s1, char* s2, char* s3, float * a, float * b, float * c, int N)
{
	char* s11; char* s22; char* s33;
	s11 = s1; s22 = s2; s33 = s3;
	while (*s11) { *s33 = *s11; s33++; s11++; }
	while (*s22) { *s33 = *s22; s33++; s22++; }
	*s33 = 0;

	hipError_t cudaStatus = mulWithCuda(a, b, c, N);
	cudaStatus = hipDeviceReset();
	return 0;
}
hipError_t mulWithCuda(float * a, float * b, float * c, int N)
{
	int size = N * sizeof(float);
	hipError_t cudaStatus;
	/*Define and initialize arrays in HOST*/
	float* h_A = (float *)malloc(size);
	float* h_B = (float *)malloc(size);
	float* h_C = (float *)malloc(size);

	for (int i = 0; i < N; i++) {
		h_A[i] = a[i];
		h_B[i] = b[i];
		h_C[i] = c[i];
	}

	/*Define and allocate arrays in DEVICE*/
	float* d_A;
	float* d_B;
	float* d_C;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	/*Copy arrays from HOST to DEVICE*/
	cudaStatus = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

	/*Define level of parallelism*/
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(1, 1, 1);

	/*Launch kernel and synchronize*/
	mulKernel << <numBlocks, threadsPerBlock >> > (d_A, d_B, d_C);
	cudaStatus = hipDeviceSynchronize();

	/*Copy output array from DEVICE to HOST*/
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		c[i] = h_C[i];
	}
	/*Free device memory*/
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return cudaStatus;
}

//����������������
__global__ void transKernel(float* inputMatrix, float* outputMatrix, int width, int height)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	for (int x = 0; x < width; x++)
		for (int y = 0; y < height; y++)
			outputMatrix[x * height + y] = inputMatrix[y * width + x];
}
int transpose(char* s1, char* s2, char* s3, float * a, float * b,  int N, int M)
{
	char* s11; char* s22; char* s33;
	s11 = s1; s22 = s2; s33 = s3;
	while (*s11) { *s33 = *s11; s33++; s11++; }
	while (*s22) { *s33 = *s22; s33++; s22++; }
	*s33 = 0;

	hipError_t cudaStatus = transWithCuda(a, b, N, M);
	cudaStatus = hipDeviceReset();
	return 0;
}
hipError_t transWithCuda(float * a, float * b, int N, int M)
{
	int width = N;     //������ �������
	int height = M;    //������ �������
	int size = width * height;

	hipError_t stat;

	//��������� ������ ��� ������� �� �����
	float* a_h = (float *)malloc(sizeof(float) * size);
	float* b_h = (float *)malloc(sizeof(float) * size);

	// define an mxk matrix a column by column
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			a_h[IDX2C(i, j, N)] = a[IDX2C(i, j, N)];
		} 
	} 

	//��������� ������ ��� ������� �� �������
	float* d_a; //�������� ������� 
	float* d_b; //����������������� ������� 

	//�������� ���������� ������ ��� ������ ������ �� �������
	hipMalloc((void**)&d_a, size * sizeof(float));
	hipMalloc((void**)&d_b, size * sizeof(float));

	//�������� �������� ������� � ����� �� ������
	hipMemcpy(d_a, a_h, size * sizeof(float), hipMemcpyHostToDevice);


	dim3 gridSize = dim3(width / 2, height / 2, 1);
	dim3 blockSize = dim3(2, 2, 1);
	if (size > 256) {
		blockSize = dim3(256, 256, 1);
		dim3 gridSize = dim3(width / 256, height / 256, 1);
	}

	//������ ���� 
	transKernel << <gridSize, blockSize >> > (d_a, d_b, width, height);


	//�������� ��������� � ������� �� ����
	hipMemcpy(b_h, d_b, size * sizeof(float), hipMemcpyDeviceToHost);


	float* C = (float *)malloc(sizeof(float) * size);

	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			C[IDX2C(i, j, M)] = b_h[IDX2C(i, j, M)];
			b[IDX2C(i, j, M)] = b_h[IDX2C(i, j, M)];
		}
	}

	//�������� �������� � ����������
	hipFree(d_a);
	hipFree(d_b);

	free(a_h);
	free(b_h);
	free(C);
	return stat;
}

//��������� ������
__global__ void matMulKernel(float *a, float *b, float *c, int M, int K, int N) {
	//������ ����� ��������� 1 ������� ������� �
	//��������� ������������ �� ������ � �� ������� �
	float Cvalue = 0.0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row > K || col > K) return;
	for (int e = 0; e < M; ++e)
		Cvalue += (a[IDX2C(row, e, M)]) * (b[IDX2C(e, col, K)]);
	c[IDX2C(row, col, M)] = Cvalue;
}
int matMul(char* s1, char* s2, char* s3, float * a, float * b, float * c, int M, int K, int N)
{
	char* s11; char* s22; char* s33;
	s11 = s1; s22 = s2; s33 = s3;
	while (*s11) { *s33 = *s11; s33++; s11++; }
	while (*s22) { *s33 = *s22; s33++; s22++; }
	*s33 = 0;

	hipError_t cudaStatus = matMulWithCuda(a, b, c, M,K,N);
	cudaStatus = hipDeviceReset();
	return 0;
}
hipError_t matMulWithCuda(float * a, float * b, float * c, int M,int K, int N)
{

	hipError_t cudaStatus;
	//����� ��� ������� �, �, � � ���
	float *h_a = (float*)malloc(M*K * sizeof(float));
	float *h_b = (float*)malloc(K*N * sizeof(float));
	float *h_c = (float*)malloc(M*N * sizeof(float));
	//����� ��� ������� �, �, � � ������ ���
	float *d_a;
	float *d_b;
	float *d_c;

	for (int i = 0; i < M; i++) 
		for (int j = 0; j < K; j++) 
			h_a[IDX2C(i, j, M)] = a[IDX2C(i, j, M)];

	for (int i = 0; i < K; i++)
		for (int j = 0; j < N; j++)
			h_b[IDX2C(i, j, K)] = b[IDX2C(i, j, K)];

	for (int i = 0; i < M; i++)
		for (int j = 0; j < N; j++)
			h_c[IDX2C(i, j, M)] = c[IDX2C(i, j, M)];


	//��������� ������ � ���
	hipMalloc((void**)&d_a, M*K * sizeof(float));
	hipMalloc((void**)&d_b, K*N * sizeof(float));
	hipMalloc((void**)&d_c, M*N * sizeof(float));
	//����������� ��������
	hipMemcpy(d_a, h_a, M*K * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, K*N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, M*N * sizeof(float), hipMemcpyHostToDevice);

	//����������� ������� ����� � ������� (block dimension, grid dimension)
	dim3 dimBlock(4, 4);
	dim3 dimGrid((K + dimBlock.x - 1) / dimBlock.x, (K + dimBlock.y - 1) / dimBlock.y);

	//����� ������-������� (������������ ���������� ��������� ������� �)
	matMulKernel << <dimGrid, dimBlock >> > (d_a, d_b, d_c, M, K, N);
	hipDeviceSynchronize();

	//����������� �������� �� ��� � ����� � ���
	hipMemcpy(h_c, d_c, M*N * sizeof(float), hipMemcpyDeviceToHost);
	//����������� �� ������ � �������� �������
	for (int i = 0; i < M; i++)
		for (int j = 0; j < N; j++)
			c[IDX2C(i, j, M)] = h_c[IDX2C(i, j, M)];
	//Free device memory
	hipFree(d_a);
	hipFree(d_b);

	return cudaStatus;
}